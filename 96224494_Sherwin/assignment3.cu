
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
using namespace std;

__global__ void AddArray(int* d_a,int col,int row)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int id=col*y+x;
    int temp;
    if(x < col &&y < row)
    temp = d_a[id];
    __syncthreads();
    int idr=row*x+y;
    d_a[idr]=temp;
   
   
}
int main()
{
    int row,col;
    printf("enter row and col");
    scanf("%d%d",&row,&col);
     
 
    int h_a[row][col],h_b[col][row];
    int Array_Bytes = row*col* sizeof(int);  
    for(int i=0; i<row; i++)
    {
       for(int j=0;j<col;j++)
       {
            h_a[i][j] = col*i+j;
            printf("%d\t", h_a[i][j]);
        }
      printf("\n");  
    }
    printf("\n");
    int *d_a;
   
    hipMalloc((void**)&d_a, Array_Bytes);
 
    // Copy the array from CPU (h_in) to the GPU (d_in)
   
    hipMemcpy(d_a, h_a, Array_Bytes, hipMemcpyHostToDevice);
    AddArray<<<dim3(col,row,1),1 >>>(d_a,col,row);
    // Copy the resulting array from GPU (d_out) to the CPU (h_out)
    hipMemcpy(h_b, d_a, Array_Bytes, hipMemcpyDeviceToHost);
    for(int i=0; i<col; i++)
    {
       for(int j=0;j<row;j++)
       {
           printf("%d\t", h_b[i][j]);
        }
       printf("\n");
    }
    hipFree(d_a);
}



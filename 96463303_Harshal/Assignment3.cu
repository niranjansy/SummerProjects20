
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int Array_Size_x,Array_Size_y;

__global__ void Sum(float* d_in1,int* d_array_size_x,int* d_array_size_y)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int k = threadIdx.y + blockIdx.y * blockDim.y;


        if (j < *d_array_size_y && k < *d_array_size_x)
        {
            int i1 = j + k * *d_array_size_y;
            int i2 = k + j * *d_array_size_x;

            float temp = d_in1[i1];
			__syncthreads();
            d_in1[i2]=temp;
        }

}
int main()
{
    cout << "Enter the array size (row , col) : ";
    cin >> Array_Size_x >> Array_Size_y;


    int Array_Bytes = Array_Size_x * sizeof(float) * Array_Size_y;

    float *h_in1,*h_out;

    h_in1 = (float*)malloc(Array_Bytes);
    h_out = (float*)malloc(Array_Bytes);

    for(int i=0; i<Array_Size_x; i++)
        for(int j = 0; j < Array_Size_y; j++)
            h_in1[i*Array_Size_y + j] = i + 0.1;

    for(int i=0; i<Array_Size_x; i++)
    {
        for(int j = 0; j < Array_Size_y; j++)
            cout << h_in1[i*Array_Size_y + j] << " ";
        cout << endl;
    }

    cout << endl;
    float *d_in1;
    int *d_array_size_x,*d_array_size_y;

    hipMalloc((void**)&d_in1, Array_Bytes);
    hipMalloc((void**)&d_array_size_x, sizeof(int));
    hipMalloc((void**)&d_array_size_y, sizeof(int));

    hipMemcpy(d_in1, h_in1, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_array_size_y, &Array_Size_y, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_array_size_x, &Array_Size_x, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil(1.0*Array_Size_y/dimBlock.x),(int)ceil(1.0*Array_Size_x/dimBlock.y));

    Sum<<<dimGrid, dimBlock>>>(d_in1,d_array_size_x,d_array_size_y);

    hipMemcpy(h_out, d_in1, Array_Bytes, hipMemcpyDeviceToHost);

    for(int i=0; i<Array_Size_y; i++)
    {
        for(int j = 0; j < Array_Size_x; j++)
            cout << h_out[i*Array_Size_x + j]<< " ";
        cout << endl;
    }

    hipFree(d_in1);
    hipFree(d_array_size_x);
    hipFree(d_array_size_y);
}

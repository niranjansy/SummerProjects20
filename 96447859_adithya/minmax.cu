
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void Min(int* d_a, int* d_b, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int t_id = threadIdx.x;
    int b_id = blockIdx.x;

	__shared__ int a[1024];

    if(id < size)
     a[t_id] = d_a[id];    

	__syncthreads();

	for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=size || id+s>=size)
            continue;
        if(t_id<s)
            {
               if(a[t_id] > a[t_id + s])
                a[t_id]= a[t_id + s];
            }
    }
    __syncthreads();

	 if(t_id==0)
        d_b[b_id] = a[t_id];   
}

__global__ void Max(int* d_a, int* d_b, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	int t_id = threadIdx.x;
    int b_id = blockIdx.x;

	__shared__ int a[1024];

    if(id < size)
     a[t_id] = d_a[id];    

	__syncthreads();

	for(int s = 512; s>0; s = s/2)
    {
        __syncthreads();
        if(id>=size || id+s>=size)
            continue;
        if(t_id<s)
            {
               if(a[t_id] < a[t_id + s])
                a[t_id] = a[t_id + s];
            }
    }
    __syncthreads();

	 if(t_id==0)
        d_b[b_id] = a[t_id];   
}

int main() 
{
    int size;
    cin>>size;
    int h_a[size], h_min, h_max;
    int bytes=size*sizeof(int);
    int length=(int)ceil(1.0*size/1024);
    for(int i=0;i<size;i++)
    {
        h_a[i]=i+1;
    }
    int *d_a, *d_b, *d_min, *d_max;
    hipMalloc((void**)&d_b, bytes);
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_min, sizeof(int));
    hipMalloc((void**)&d_max, sizeof(int));
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    
        Min<<<((int)ceil(1.0*size/1024)), 1024>>>(d_a, d_b, size);
	Min<<<1, 1024>>>(d_b, d_min, length);
	
	Max<<<((int)ceil(1.0*size/1024)), 1024>>>(d_a, d_b, size);
	Max<<<1, 1024>>>(d_b, d_max, length);
	
	hipMemcpy(&h_min, d_min, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);

	int min=h_a[0], max=h_a[0];
	for(int i=1;i<size;i++)
	{
	    if(h_a[i]<min)
		min=h_a[i];
	    if(h_a[i]>max)
         	max=h_a[i];
	}

	if(h_min==min && h_max==max)
	cout<<"correct result";
	else
        cout<<"Invalid";
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_max);
	hipFree(d_min);
}
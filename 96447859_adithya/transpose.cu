
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void transpose(int *d_a, int *d_b, int r, int c)
{
    int x=blockIdx.x*blockDim.x + threadIdx.x;
    int y=blockIdx.y*blockDim.y + threadIdx.y;
    
    if(x<c && y<r)
    {
        int index_1=c*y+x;
        int index_2=r*x+y;
        int temp=d_a[index_1];
        __syncthreads();
        d_b[index_2]=temp;
    }
}

int main() 
{
    int r,c;
    cin>>r>>c;
    int bytes=r*c*sizeof(int);
    int h_a[r][c], h_b[c][r];
    for(int i=0;i<r;i++)
    {
        for(int j=0;j<c;j++)
        {
	        h_a[i][j]=i*(j+1);
        }
    }
    
    int *d_a, *d_b;
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    transpose<<<dim3(c,r,1),1 >>>(d_a,d_b,r,c);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    
    for(int i=0; i<c; i++)
    {
        for(int j=0; j<r; j++)
        {
           cout<<h_b[i][j];
        }
        cout<<"\n";
    }
    hipFree(d_a);
    hipFree(d_b);
}
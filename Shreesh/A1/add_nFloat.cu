#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<cstdlib>
#include<ctime>
#include<math.h>
#include<chrono>
using namespace std;
using namespace std::chrono;

//Global variable 
unsigned long long int size;

//Display logged array
void showArr(float* arr, char x){
    cout << "\n\nArray " << x << " logged: " << endl;
    for(int i = 0; i < size; i++){
        cout << arr[i] << ", ";
    }
}

//Random number filler 
void fillRandom(float* arr){
    srand((unsigned int)time(NULL));
    for(int i = 0; i < size; i++){
        float random = (float(rand())/float((RAND_MAX)))*10;
        random = (float)(((int)(random * 100))/100.0);
        arr[i] = random;
    }
}

//Compare arrays
void compArrs(float* a, float*b){
    bool same = true;
    for(int i = 0; i < size; i++){
        if(a[i] != b[i]){
            same = false;
            break;
        }
    }

    if(same){
        cout << "Comparision successful!" << endl;
    }
    else{
        cout << "Comparision unsuccessful :/" << endl;
    }
}

//Function: CPU
void cpu_add(float* h_a, float* h_b, float* h_d){
    for (int i = 0; i < size; i++)
    {
        h_d[i] = h_a[i] + h_b[i];
    }
}

//Kernel: GPU
__global__ void Add(float* d_a, float* d_b, float* d_c){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    d_c[id] = d_a[id] + d_b[id];
}


int main(){
    //Init GPU pointers
    float *d_a = NULL;
    float *d_b = NULL;
    float *d_c = NULL;

    cout << "Enter number of elements: "; cin >> size;
    //Init input arrays
    float h_a[size], h_b[size], h_c[size], h_d[size];
    
    //Log arrays
    string s = "random";
    cout << "\nRandom number generator or User input? (r/u): ";
    cin >> s; 
    
    //Choice of Random number generator or user input
    if(s=="u"){
        cout << "\nEnter " << size << " numbers for array A " << endl;
        for(int i = 0; i < size; i++){
            cin >> h_a[i];
        }
        cout << "\nEnter " << size << " numbers for array B " << endl;
        for(int i = 0; i < size; i++){
            cin >> h_b[i];
        }
    }
    else{
        fillRandom(h_a);
        fillRandom(h_b);
    }

    //Display arrays
    if(size > 10){
        cout << "\nArray size too large, do you still want me to display?(y/n): ";
        cin >> s; 
        if(s == "y"){
            showArr(h_a, 'A');
            showArr(h_b, 'B');
        }
    }
    else{
        showArr(h_a, 'A');
        showArr(h_b, 'B');
    }


    // Computing using CPU
    //start time stamp 
    auto start_cpu = high_resolution_clock::now();
    cpu_add(h_a, h_b, h_d);
    //stop time stamp
    auto stop_cpu = high_resolution_clock::now();
    auto cpu_time = duration_cast<nanoseconds>(stop_cpu-start_cpu);

    //CPU result
    if(s != "n"){ //dont display if the array size is too large
        cout << "\n\nCPU Result" << endl;
        for (int i = 0; i < size; i++)
        {
            cout << h_d[i] << ", ";
        }
    }

    //Computing using GPU    
    //Allocating memory in GPU pointed by d_x (x=a,b,c)
    int arr_bytes = size * sizeof(float);
    hipMalloc((void**)&d_a, arr_bytes);
    hipMalloc((void**)&d_b, arr_bytes);
    hipMalloc((void**)&d_c, arr_bytes);

    // Copying CPU -->  GPU memory
    hipMemcpy(d_a, h_a, arr_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arr_bytes, hipMemcpyHostToDevice);

    //Kernel call; Block: b, Threads: 1024 (max possible)
    //b = upper_ceil(1.0*size/1024)
    
    //time start stamp
    auto start_gpu = high_resolution_clock::now();
    Add<<< ceil(1.0*size/1024), 1024 >>>(d_a, d_b, d_c);
    //time end stamp 
    auto stop_gpu = high_resolution_clock::now();
    auto gpu_time = duration_cast<nanoseconds>(stop_gpu-start_gpu);

    //Copying GPU --> CPU memory
    hipMemcpy(h_c, d_c, arr_bytes, hipMemcpyDeviceToHost);

    //GPU Result 
    if(s != "n"){ //dont display if the array size is too large
        cout << "\n\nGPU Result" << endl;
        for(int i = 0;i < size; i++){
            cout << h_c[i] << ", ";
        }
    }

    //De-allocating memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c); 

    //Success if all elements match
    cout << "\nResult analysis: " << endl;
    compArrs(h_c, h_d);

    //Execution time analysis; 
    //GPU yields faster results for typically large array sizes 
    cout << "\nTime analysis:" << endl;
    cout << "CPU execution time: " << cpu_time.count() << " nanosec" << endl;
    cout << "GPU execution time: " << gpu_time.count() << " nanosec" << endl;
}
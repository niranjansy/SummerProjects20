
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include<iomanip>
#include<chrono>
using namespace std;
using namespace std::chrono;

//Device code
__global__ void AddN(float* a, float* b, 
    float* c, int* ro, int* co){
    int x, y, idx;
    //assign a thread to each element (i,j)
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    
    /*
    Addressing logic:
    Traverse y number of column lengths followed 
    by an offset of x
    */

    idx = y * (*co) + x;
    
    if(y < *ro && x < *co){
        *(c+idx) = *(a+idx) + *(b+idx);
        // c[idx] = a[idx] + b[idx];
    }
    }

//Host code
int row, col;

void printArr(float *arr, char x);
void fillRandom(float* arr, unsigned int seed);
void cpu_add(float* a, float* b, float* d);
void resultComp(float* a, float* b);

int main() 
{ 
    cout << "Enter row and col for the matrices: " << endl;
    cout << "Row: "; cin >> row;
    cout << "Column: "; cin >> col;

    char display = 'n';
    if(col <= 3)
    display = 'y';
    else
    display = 'n';

    if(col > 3){
        cout << "Do you want to display the results?(y/n): "; 
        cin >> display;
    }
	float h_a[row][col]; //array A
	float h_b[row][col]; //array B
	float h_c[row][col]; //gpu result
	float h_d[row][col]; //cpu result

    //fill arrays with random floats 
    fillRandom((float *)h_a, 1);
    fillRandom((float *)h_b, 0);

    //Using CPU
    // cout << "Adding using CPU:" << endl;
    auto start_cpu = high_resolution_clock::now();
    cpu_add((float *)h_a, (float *)h_b, (float *)h_d);
    //stop time stamp
    auto stop_cpu = high_resolution_clock::now();
    auto cpu_time = duration_cast<nanoseconds>(stop_cpu-start_cpu);

    if(display=='y'){
        cout << "\nCPU Result: ";
        printArr((float *)h_d, 'D');
    }

    //Using GPU 
    //Pointers for GPU memory
    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;
    int* co = NULL;
    int* ro = NULL;
    //Allocating GPU memory
    int array_bytes = col * row * sizeof(float);
    hipMalloc((void**)&d_a, array_bytes);
    hipMalloc((void**)&d_b, array_bytes);
    hipMalloc((void**)&d_c, array_bytes);
    hipMalloc((void**)&co, sizeof(int));
    hipMalloc((void**)&ro, sizeof(int));

    //Copying CPU --> GPU memory
    hipMemcpy(d_a, h_a, array_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, array_bytes, hipMemcpyHostToDevice);
    hipMemcpy(ro, &row, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(co, &col, sizeof(int), hipMemcpyHostToDevice);

    int bx, by, tx, ty;
    tx = ty = 32;
    bx = (int)ceil(1.0*col/tx);
    by = (int)ceil(1.0*row/ty);
    dim3 dimGrid(bx, by);
    dim3 dimBlock(tx, ty);

    //Kernel call
    // cout << "\nAdding using GPU:" << endl;
    auto start_gpu = high_resolution_clock::now();
    AddN<<< dimGrid, dimBlock >>> (d_a, d_b, d_c, ro, co);
    //time end stamp 
    auto stop_gpu = high_resolution_clock::now();
    auto gpu_time = duration_cast<nanoseconds>(stop_gpu-start_gpu);


    //Copying GPU --> CPU memory
    hipMemcpy(h_c, d_c, array_bytes, hipMemcpyDeviceToHost);
    if(display=='y'){
        cout << "\nGPU result:";
        printArr((float *)h_c, 'C');
    }

    resultComp((float*)h_c, (float*)h_d);
    cout << "\nTime analysis:" << endl;
    cout << "CPU execution time: " << cpu_time.count() << " nanosec" << endl;
    cout << "GPU execution time: " << gpu_time.count() << " nanosec" << endl;

    //De-allocate GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(ro);
    hipFree(co);

    return 0; 
} 

void printArr(float *arr, char x)
{ 
    cout << "\n\nArray " << x << ": " << endl;
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            float val = *((arr+i*col)+j);
            cout << fixed << setprecision(2) << val << "  ";
        }
        cout << endl;
    }
} 

void fillRandom(float *arr, unsigned int seed){
    srand((unsigned int)seed);
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            //generate random number
            float num = rand();
            float max = RAND_MAX;
            float random = num / max;
            random = int(random * 1000.0); 
            random = random / 100.0;
            //update
            *((arr + i*col)+j) = random;
        }
    }
}

void cpu_add(float* a, float* b, float* d){
    for (int y = 0; y < row; y++)
    {
        for (int x = 0; x < col; x++)
        {
            *((d+y*col)+x) = *((a+y*col)+x) + *((b+y*col)+x);
        }
    }
    
}

void resultComp(float* a, float* b){
    cout << "\nResult Analysis: " << endl;
    bool same = true;
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            float val1 = *(a+i*col+j);
            float val2 = *(b+i*col+j); 
            if( val1 != val2 ){
                same = false;
                break;
            }
        }
        
    }

    if(same){
        cout << "Success!" << endl;
    }
    else
    {
        cout << "Result doesn't match :/" << endl;
    }
    cout << "\n[Note: CPU will perform better for smaller mat sizes]" << endl;
        
}


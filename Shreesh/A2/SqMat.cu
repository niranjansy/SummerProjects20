#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
using namespace std;

//Device code
__global__ void AddN(float* a, float* b, 
    float* c, int* s){
    int i, j, idx;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    
    idx = i + (*s) * j;
    
    if(j < *s && i < *s){
        c[idx] = a[idx] + b[idx];
    }
    }

//Host code
int size;

void printArr(float *arr, char x);
void fillRandom(float* arr, unsigned int seed);
void cpu_add(float* a, float* b, float* d);
void resultComp(float* a, float* b);

int main() 
{ 
    cout << "Enter size of the square matrices: ";
    cin >> size;
    char display = 'n';

    if(size <= 3)
    display = 'y';
    else
    display = 'n';

    if(size > 3){
        cout << "Do you want to display the results?(y/n): "; 
        cin >> display;
    }
	float h_a[size][size]; //array A
	float h_b[size][size]; //array B
	float h_c[size][size]; //gpu result
	float h_d[size][size]; //cpu result

    //fill arrays with random floats 
    fillRandom((float *)h_a, 1);
    fillRandom((float *)h_b, 0);

    //Using CPU
    // cout << "Adding using CPU:" << endl;
    cpu_add((float *)h_a, (float *)h_b, (float *)h_d);

    if(display=='y'){
        cout << "\nCPU Result: ";
        printArr((float *)h_d, 'D');
    }

    //Using GPU 
    //Pointers for GPU memory
    float* d_a = NULL;
    float* d_b = NULL;
    float* d_c = NULL;
    int* s = NULL;
    //Allocating GPU memory
    int array_bytes = size * size * sizeof(float);
    hipMalloc((void**)&d_a, array_bytes);
    hipMalloc((void**)&d_b, array_bytes);
    hipMalloc((void**)&d_c, array_bytes);
    hipMalloc((void**)&s, sizeof(int));

    //Copying CPU --> GPU memory
    hipMemcpy(d_a, h_a, array_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, array_bytes, hipMemcpyHostToDevice);
    hipMemcpy(s, &size, sizeof(int), hipMemcpyHostToDevice);

    /*
        Kernel call; Block: b, Threads: t (max possible)
        n = size;
    - Since the maximum number of threads per block is limited to 1024, 
        we make squares of length sqrt(1024) = 32 
            t = 32, b = upper_ceil(n/32)
            parameter: dim(b,b,1), dim3(t,t,1)
    */      

    int bx, by, tx, ty;
    tx = ty = 32;
    bx = by = (int)ceil(1.0*size/32);       

    dim3 dimGrid(bx, by);   
    dim3 dimBlock(tx, ty);  

    //Kernel call
    // cout << "\nAdding using GPU:" << endl;
    AddN<<< dimGrid, dimBlock >>> (d_a, d_b, d_c, s);

    //Copying GPU --> CPU memory
    hipMemcpy(h_c, d_c, array_bytes, hipMemcpyDeviceToHost);
    if(display=='y'){
        cout << "\nGPU result:";
        printArr((float *)h_c, 'C');
    }

    resultComp((float*)h_c, (float*)h_d);

    //De-allocate GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(s);

    return 0; 
} 

void printArr(float *arr, char x)
{ 
    cout << "\n\nArray " << x << ": " << endl;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            cout << *((arr+i*size)+j) << "  ";
        }
        cout << endl;
    }
} 

void fillRandom(float *arr, unsigned int seed){
    srand((unsigned int)seed);
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            //generate random number
            float num = rand();
            float max = RAND_MAX;
            float random = num / max;
            random = int(random * 1000.0); 
            random = random / 100.0;
            //update
            *((arr + i*size)+j) = random;
        }
    }
}

void cpu_add(float* a, float* b, float* d){
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            *((d+i*size)+j) = *((a+i*size)+j) + *((b+i*size)+j);
        }
    }
    
}

void resultComp(float* a, float* b){
    bool same = true;
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            float val1 = *(a+i*size+j);
            float val2 = *(b+i*size+j); 
            if( val1 != val2 ){
                same = false;
                break;
            }
        }
        
    }

    if(same){
        cout << "Success!" << endl;
    }
    else
    {
        cout << "Result doesn't match" << endl;
    }
    
        
}


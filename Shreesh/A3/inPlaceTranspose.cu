        
#include <hip/hip_runtime.h>
#include<iostream>
        using namespace std;

        //Device code
        __global__ void Transpose(int* d_arr, int *r, int *c){

            int x = blockDim.x * blockIdx.x + threadIdx.x;
            int y = blockDim.y * blockIdx.y + threadIdx.y;  
            __syncthreads();

            if(x < *c && y < *r){
                int idx_o = x + (*c) * y;
                // int idx_n = x * (*r) + y;
                //alternatively
                int idx_n = idx_o * (*r) - ((*r)*(*c)-1)*(y);
                // printf("Sending %d to %d \n", idx_o, idx_n);
                __syncthreads();

                //Read  
                int temp = d_arr[idx_o];

                __syncthreads();

                //Write
                d_arr[idx_n] = temp;
            }
        }


        //Host code
        void showArray(int* arr, int row, int col);
        void fillArray(int* arr, int row, int col);

        int main(int argc, char const *argv[])
        {
            //CPU fields
            int row, col;
            cout << "Enter dimensions of matrix: " << endl;
            cout << "Row: "; cin >> row; 
            cout << "Col: "; cin >> col; cout << endl;
            int array_bytes = row * col * sizeof(int);
            int* h_arr = (int*)malloc(array_bytes);

            //GPU fields 
            int *d_arr = NULL;
            int *r = NULL;
            int *c = NULL;
            hipMalloc((void**)&d_arr, array_bytes);
            hipMalloc((void**)&r, sizeof(int));
            hipMalloc((void**)&c, sizeof(int));

            //Fill  
            fillArray((int*)h_arr, row, col);

            //Print input
            cout << "Input Matrix: " << endl;
            showArray((int*)h_arr, row, col);

            //Tranpose
            //Copying CPU --> GPU memory
            hipMemcpy(d_arr, h_arr, array_bytes, hipMemcpyHostToDevice);
            hipMemcpy(r, &row, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(c, &col, sizeof(int), hipMemcpyHostToDevice);

            dim3 dimBlock(32, 32);
            dim3 dimGrid((int)ceil(1.0*col/32), (int)ceil(1.0*row/32));

            Transpose<<<dimGrid, dimBlock>>>(d_arr, r, c);

            //Copying GPU --> CPU memory
            hipMemcpy(h_arr, d_arr, array_bytes, hipMemcpyDeviceToHost);
            hipMemcpy(&row, c, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&col, r, sizeof(int), hipMemcpyDeviceToHost);

            //Print output
            cout << "Output Matrix" << endl;
            showArray((int*)h_arr, row, col);

            hipFree(d_arr);
            hipFree(c);
            hipFree(r);

            return 0;
        }

        void showArray(int* arr, int row, int col){
            for (int i = 0; i < row; i++)
            {
                for (int j = 0; j < col; j++)
                {
                    cout << *((arr+i*col+j)) << " ";
                }
                cout << endl;
            }
        }

        void fillArray(int* arr, int row, int col){
            int count = 0;
            for (int i = 0; i < row; i++)
            {
                for (int j = 0; j < col; j++)
                {
                    *((arr+i*col+j)) = ++count;
                }
                
            }
        }